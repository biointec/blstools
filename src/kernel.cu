
#include <hip/hip_runtime.h>
__global__
void filterScore(float *R, int m, int n, float *threshold, int* occIdx, float* occScore, int *nOcc)
{
  	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = j*m + i; // column-major storage

  	if (i < m && j < n && R[idx] >= threshold[i]) { // FIXME: consistency with other filters (threshold)
		int resPos = atomicAdd(nOcc, 1);
		occScore[resPos] = R[idx];
		occIdx[resPos] = idx;
	}
}

void kernel_wrapper(float *d_R, int m, int n, float *d_threshold, int* d_occIdx, float* d_occScore, int *d_nOcc)
{
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

	filterScore<<<numBlocks, threadsPerBlock>>>(d_R, m, n, d_threshold, d_occIdx, d_occScore, d_nOcc);
}
